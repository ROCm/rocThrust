#include "hip/hip_runtime.h"
// MIT License
//
// Copyright (c) 2024 Advanced Micro Devices, Inc. All rights reserved.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

// Benchmark utils
#include "../../../bench_utils/bench_utils.hpp"

// rocThrust
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>

// Google Benchmark
#include <benchmark/benchmark.h>

// STL
#include <cstdlib>
#include <string>
#include <vector>

struct _max
{
    template <typename T, typename Policy = thrust::detail::device_t>
    float64_t run(thrust::device_vector<T> input, thrust::device_vector<T> output)
    {
        bench_utils::gpu_timer d_timer;

        d_timer.start(0);
        thrust::inclusive_scan(
            Policy {}, input.cbegin(), input.cend(), output.begin(), bench_utils::max_t {});
        d_timer.stop(0);

        return d_timer.get_duration();
        return 0;
    }
};

template <class Benchmark, class T>
void run_benchmark(benchmark::State& state, const std::size_t elements, const std::string seed_type)
{
    // Benchmark object
    Benchmark benchmark {};

    // GPU times
    std::vector<double> gpu_times;

    // Generate input
    thrust::device_vector<T> input = bench_utils::generate(elements, seed_type);

    // Output
    thrust::device_vector<T> output(elements);

    for(auto _ : state)
    {
        float64_t duration = benchmark.template run<T>(input, output);
        state.SetIterationTime(duration);
        gpu_times.push_back(duration);
    }

    // BytesProcessed include read and written bytes, so when the BytesProcessed/s are reported
    // it will actually be the global memory bandwidth gotten.
    state.SetBytesProcessed(state.iterations() * 2 * elements * sizeof(T));
    state.SetItemsProcessed(state.iterations() * elements);

    const double gpu_cv         = bench_utils::StatisticsCV(gpu_times);
    state.counters["gpu_noise"] = gpu_cv;
}

#define CREATE_BENCHMARK(T, Elements)                                                        \
    benchmark::RegisterBenchmark(                                                            \
        bench_utils::bench_naming::format_name("{algo:inclusive_scan,subalgo:" + name        \
                                               + ",input_type:" #T + ",elements:" #Elements) \
            .c_str(),                                                                        \
        run_benchmark<Benchmark, T>,                                                         \
        Elements,                                                                            \
        seed_type)

#define BENCHMARK_TYPE(type)                                          \
    CREATE_BENCHMARK(type, 1 << 16), CREATE_BENCHMARK(type, 1 << 20), \
        CREATE_BENCHMARK(type, 1 << 24), CREATE_BENCHMARK(type, 1 << 28)

template <class Benchmark>
void add_benchmarks(const std::string&                            name,
                    std::vector<benchmark::internal::Benchmark*>& benchmarks,
                    const std::string                             seed_type)
{
    std::vector<benchmark::internal::Benchmark*> bs
        = { BENCHMARK_TYPE(int8_t),
            BENCHMARK_TYPE(int16_t),
            BENCHMARK_TYPE(int32_t),
            BENCHMARK_TYPE(int64_t)
#if THRUST_BENCHMARKS_HAVE_INT128_SUPPORT
                ,
            BENCHMARK_TYPE(int128_t)
#endif
                ,
            BENCHMARK_TYPE(float32_t),
            BENCHMARK_TYPE(float64_t) };
    benchmarks.insert(benchmarks.end(), bs.begin(), bs.end());
}

int main(int argc, char* argv[])
{
    cli::Parser parser(argc, argv);
    parser.set_optional<std::string>(
        "name_format", "name_format", "human", "either: json,human,txt");
    parser.set_optional<std::string>("seed", "seed", "random", bench_utils::get_seed_message());
    parser.run_and_exit_if_error();

    // Parse argv
    benchmark::Initialize(&argc, argv);
    bench_utils::bench_naming::set_format(
        parser.get<std::string>("name_format")); /* either: json,human,txt */
    const std::string seed_type = parser.get<std::string>("seed");

    // Benchmark info
    bench_utils::add_common_benchmark_info();
    benchmark::AddCustomContext("seed", seed_type);

    // Add benchmark
    std::vector<benchmark::internal::Benchmark*> benchmarks;
    add_benchmarks<_max>("max", benchmarks, seed_type);

    // Use manual timing
    for(auto& b : benchmarks)
    {
        b->UseManualTime();
        b->Unit(benchmark::kMicrosecond);
        b->MinTime(0.4); // in seconds
    }

    // Run benchmarks
    benchmark::RunSpecifiedBenchmarks(new bench_utils::CustomReporter);

    // Finish
    benchmark::Shutdown();
    return 0;
}
