// MIT License
//
// Copyright (c) 2024 Advanced Micro Devices, Inc. All rights reserved.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

// Benchmark utils
#include "../../../bench_utils/bench_utils.hpp"

// rocThrust
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>

// Google Benchmark
#include <benchmark/benchmark.h>

// STL
#include <cstdlib>
#include <string>
#include <vector>

struct by_key
{
    template <typename KeyT, typename ValueT, typename Policy>
    float64_t run(thrust::device_vector<KeyT>&   input_keys,
                  thrust::device_vector<ValueT>& input_vals,
                  thrust::device_vector<ValueT>& output_vals,
                  Policy                         policy)
    {
        bench_utils::gpu_timer d_timer;

        d_timer.start(0);
        thrust::exclusive_scan_by_key(policy,
                                      input_keys.cbegin(),
                                      input_keys.cend(),
                                      input_vals.cbegin(),
                                      output_vals.begin());
        d_timer.stop(0);

        return d_timer.get_duration();
    }
};

template <class Benchmark, class KeyT, class ValueT>
void run_benchmark(benchmark::State& state, const std::size_t elements, const std::string seed_type)
{
    // Benchmark object
    Benchmark benchmark {};

    // GPU times
    std::vector<double> gpu_times;

    // Generate input
    thrust::device_vector<KeyT> input_keys = bench_utils::generate.uniform.key_segments(
        elements, seed_type, 0, 5200 /*magic numbers in thrust*/);
    thrust::device_vector<ValueT> input_vals(elements);

    // Output
    thrust::device_vector<ValueT> output_vals(elements);

    bench_utils::caching_allocator_t alloc {};
    thrust::detail::device_t         policy {};

    for(auto _ : state)
    {
        float64_t duration = benchmark.template run<KeyT, ValueT>(
            input_keys, input_vals, output_vals, policy(alloc));
        state.SetIterationTime(duration);
        gpu_times.push_back(duration);
    }

    // BytesProcessed include read and written bytes, so when the BytesProcessed/s are reported
    // it will actually be the global memory bandwidth gotten.
    state.SetBytesProcessed(state.iterations() * (elements * (sizeof(KeyT) + 2 * sizeof(ValueT))));
    state.SetItemsProcessed(state.iterations() * elements);

    const double gpu_cv         = bench_utils::StatisticsCV(gpu_times);
    state.counters["gpu_noise"] = gpu_cv;
}

#define CREATE_BENCHMARK(KeyT, ValueT, Elements)                                                 \
    benchmark::RegisterBenchmark(bench_utils::bench_naming::format_name(                         \
                                     "{algo:exclusive_scan,subalgo:" + name + ",key_type:" #KeyT \
                                     + ",value_type:" #ValueT + ",elements:" #Elements)          \
                                     .c_str(),                                                   \
                                 run_benchmark<Benchmark, KeyT, ValueT>,                         \
                                 Elements,                                                       \
                                 seed_type)

#define BENCHMARK_VALUE_TYPE(key_type, value_type)       \
    CREATE_BENCHMARK(key_type, value_type, 1 << 16),     \
        CREATE_BENCHMARK(key_type, value_type, 1 << 20), \
        CREATE_BENCHMARK(key_type, value_type, 1 << 24), \
        CREATE_BENCHMARK(key_type, value_type, 1 << 28)

#if THRUST_BENCHMARKS_HAVE_INT128_SUPPORT
#define BENCHMARK_KEY_TYPE(key_type)                                                      \
    BENCHMARK_VALUE_TYPE(key_type, int8_t), BENCHMARK_VALUE_TYPE(key_type, int16_t),      \
        BENCHMARK_VALUE_TYPE(key_type, int32_t), BENCHMARK_VALUE_TYPE(key_type, int64_t), \
        BENCHMARK_VALUE_TYPE(key_type, int64_t), BENCHMARK_VALUE_TYPE(key_type, int128_t)
#else
#define BENCHMARK_KEY_TYPE(key_type)                                                      \
    BENCHMARK_VALUE_TYPE(key_type, int8_t), BENCHMARK_VALUE_TYPE(key_type, int16_t),      \
        BENCHMARK_VALUE_TYPE(key_type, int32_t), BENCHMARK_VALUE_TYPE(key_type, int64_t), \
        BENCHMARK_VALUE_TYPE(key_type, int64_t)
#endif

template <class Benchmark>
void add_benchmarks(const std::string&                            name,
                    std::vector<benchmark::internal::Benchmark*>& benchmarks,
                    const std::string                             seed_type)
{
    std::vector<benchmark::internal::Benchmark*> bs
        = { BENCHMARK_KEY_TYPE(int8_t),
            BENCHMARK_KEY_TYPE(int16_t),
            BENCHMARK_KEY_TYPE(int32_t),
            BENCHMARK_KEY_TYPE(int64_t)
#if THRUST_BENCHMARKS_HAVE_INT128_SUPPORT
                ,
            BENCHMARK_KEY_TYPE(int128_t)
#endif
                ,
            BENCHMARK_KEY_TYPE(float32_t),
            BENCHMARK_KEY_TYPE(float64_t) };
    benchmarks.insert(benchmarks.end(), bs.begin(), bs.end());
}

int main(int argc, char* argv[])
{
    cli::Parser parser(argc, argv);
    parser.set_optional<std::string>(
        "name_format", "name_format", "human", "either: json,human,txt");
    parser.set_optional<std::string>("seed", "seed", "random", bench_utils::get_seed_message());
    parser.run_and_exit_if_error();

    // Parse argv
    benchmark::Initialize(&argc, argv);
    bench_utils::bench_naming::set_format(
        parser.get<std::string>("name_format")); /* either: json,human,txt */
    const std::string seed_type = parser.get<std::string>("seed");

    // Benchmark info
    bench_utils::add_common_benchmark_info();
    benchmark::AddCustomContext("seed", seed_type);

    // Add benchmark
    std::vector<benchmark::internal::Benchmark*> benchmarks;
    add_benchmarks<by_key>("by_key", benchmarks, seed_type);

    // Use manual timing
    for(auto& b : benchmarks)
    {
        b->UseManualTime();
        b->Unit(benchmark::kMicrosecond);
        b->MinTime(0.4); // in seconds
    }

    // Run benchmarks
    benchmark::RunSpecifiedBenchmarks(new bench_utils::CustomReporter);

    // Finish
    benchmark::Shutdown();
    return 0;
}
