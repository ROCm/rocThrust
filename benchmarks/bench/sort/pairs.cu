// MIT License
//
// Copyright (c) 2024 Advanced Micro Devices, Inc. All rights reserved.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

// Benchmark utils
#include "../../bench_utils/bench_utils.hpp"

// rocThrust
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>

// Google Benchmark
#include <benchmark/benchmark.h>

// STL
#include <cstdlib>
#include <string>
#include <vector>

struct pairs
{
    template <typename KeyT, typename ValueT, typename Policy = thrust::detail::device_t>
    float64_t run(thrust::device_vector<KeyT> keys, thrust::device_vector<ValueT> vals)
    {
        bench_utils::gpu_timer d_timer;

        d_timer.start(0);
        thrust::sort_by_key(Policy {}, keys.begin(), keys.end(), vals.begin());
        d_timer.stop(0);

        return d_timer.get_duration();
    }
};

template <class Benchmark, class KeyT, class ValueT>
void run_benchmark(benchmark::State& state,
                   const std::size_t elements,
                   const std::string seed_type,
                   const int         entropy_reduction)
{
    // Benchmark object
    Benchmark benchmark {};

    // GPU times
    std::vector<double> gpu_times;

    // Generate input
    const auto entropy = bench_utils::get_entropy_percentage(entropy_reduction) / 100.0f;
    thrust::device_vector<KeyT>   keys = bench_utils::generate(elements, seed_type, entropy);
    thrust::device_vector<ValueT> vals = bench_utils::generate(elements, seed_type);

    for(auto _ : state)
    {
        float64_t duration = benchmark.template run<KeyT, ValueT>(keys, vals);
        state.SetIterationTime(duration);
        gpu_times.push_back(duration);
    }

    // BytesProcessed include read and written bytes, so when the BytesProcessed/s are reported
    // it will actually be the global memory bandwidth gotten.
    state.SetBytesProcessed(state.iterations() * 2 * elements * (sizeof(KeyT) + sizeof(ValueT)));
    state.SetItemsProcessed(state.iterations() * elements);

    const double gpu_cv         = bench_utils::StatisticsCV(gpu_times);
    state.counters["gpu_noise"] = gpu_cv;
}

#define CREATE_BENCHMARK(KeyT, ValueT, Elements, EntropyReduction)                                 \
    benchmark::RegisterBenchmark(                                                                  \
        bench_utils::bench_naming::format_name(                                                    \
            "{algo:sort,subalgo:" + name + ",key_type:" #KeyT + ",value_type:" #ValueT             \
            + ",elements:" #Elements                                                               \
            + ",entropy:" + std::to_string(bench_utils::get_entropy_percentage(EntropyReduction))) \
            .c_str(),                                                                              \
        run_benchmark<Benchmark, KeyT, ValueT>,                                                    \
        Elements,                                                                                  \
        seed_type,                                                                                 \
        EntropyReduction)

#define BENCHMARK_VALUE_TYPE(key_type, value_type, entropy)       \
    CREATE_BENCHMARK(key_type, value_type, 1 << 16, entropy),     \
        CREATE_BENCHMARK(key_type, value_type, 1 << 20, entropy), \
        CREATE_BENCHMARK(key_type, value_type, 1 << 24, entropy), \
        CREATE_BENCHMARK(key_type, value_type, 1 << 28, entropy)

#define BENCHMARK_KEY_TYPE_ENTROPY(key_type, entropy)     \
    BENCHMARK_VALUE_TYPE(key_type, int8_t, entropy),      \
        BENCHMARK_VALUE_TYPE(key_type, int16_t, entropy), \
        BENCHMARK_VALUE_TYPE(key_type, int32_t, entropy), \
        BENCHMARK_VALUE_TYPE(key_type, int64_t, entropy)

template <class Benchmark>
void add_benchmarks(const std::string&                            name,
                    std::vector<benchmark::internal::Benchmark*>& benchmarks,
                    const std::string                             seed_type)
{
    constexpr int entropy_reductions[] = {0, 2, 6}; // 1.000, 0.544, 0.000;

    for(int entropy_reduction : entropy_reductions)
    {
        std::vector<benchmark::internal::Benchmark*> bs
            = {BENCHMARK_KEY_TYPE_ENTROPY(int8_t, entropy_reduction),
               BENCHMARK_KEY_TYPE_ENTROPY(int16_t, entropy_reduction),
               BENCHMARK_KEY_TYPE_ENTROPY(int32_t, entropy_reduction),
               BENCHMARK_KEY_TYPE_ENTROPY(int64_t, entropy_reduction)};
        benchmarks.insert(benchmarks.end(), bs.begin(), bs.end());
    }
}

int main(int argc, char* argv[])
{
    cli::Parser parser(argc, argv);
    parser.set_optional<std::string>(
        "name_format", "name_format", "human", "either: json,human,txt");
    parser.set_optional<std::string>("seed", "seed", "random", bench_utils::get_seed_message());
    parser.run_and_exit_if_error();

    // Parse argv
    benchmark::Initialize(&argc, argv);
    bench_utils::bench_naming::set_format(
        parser.get<std::string>("name_format")); /* either: json,human,txt */
    const std::string seed_type = parser.get<std::string>("seed");

    // Benchmark info
    bench_utils::add_common_benchmark_info();
    benchmark::AddCustomContext("seed", seed_type);

    // Add benchmark
    std::vector<benchmark::internal::Benchmark*> benchmarks;
    add_benchmarks<pairs>("pairs", benchmarks, seed_type);

    // Use manual timing
    for(auto& b : benchmarks)
    {
        b->UseManualTime();
        b->Unit(benchmark::kMicrosecond);
        b->MinTime(0.4); // in seconds
    }

    // Run benchmarks
    benchmark::RunSpecifiedBenchmarks(new bench_utils::CustomReporter);

    // Finish
    benchmark::Shutdown();
    return 0;
}
